/**
 * @file main.cu
 * @brief Main entry point for benchmarking different SpMV implementations.
 *
 * @details
 * This file serves as the primary driver for the SpMV benchmark suite:
 * - Parses command-line arguments to determine benchmark mode and matrix file
 * - Loads sparse matrices from Matrix Market (.mtx) format files
 * - Selects and initializes the appropriate SpMV operator (CSR, ELLPACK, or STENCIL)
 * - Allocates and initializes input/output vectors on host memory
 * - Executes the SpMV benchmark using the selected GPU implementation
 * - Manages memory cleanup and resource deallocation
 * 
 * The program supports three SpMV implementations:
 * - CSR (Compressed Sparse Row) format
 * - ELLPACK format for regular sparsity patterns
 * - STENCIL format for structured grid operations
 *
 * Author: Bouhrour Stephane
 * Date: 2025-07-15
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "spmv.h"
#include "io.h"
#include "benchmark_stats.h"

/**
 * @brief Main function - Entry point for SpMV benchmark program.
 * @details Orchestrates the entire benchmark workflow including argument parsing,
 * matrix loading, operator selection, memory management, and benchmark execution.
 * The function expects command-line arguments specifying the matrix file and
 * the desired SpMV implementation mode.
 * 
 * Expected usage: ./program <matrix_file.mtx> --mode=<csr|ellpack|stencil>
 * 
 * @param argc Number of command-line arguments
 * @param argv Array of command-line argument strings
 * @return EXIT_SUCCESS (0) on successful completion, EXIT_FAILURE (1) on error
 */
int main(int argc, char* argv[]) {
    // Check for correct number of command-line arguments
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <matrix_file.mtx> --mode=<mode1[,mode2,...]> [--output-format=<human|json|csv>] [--output-file=<filename>]\n", argv[0]);
        fprintf(stderr, "Available modes: csr, ellpack-naive, stencil5, stencil5-opt, stencil5-shared, stencil5-coarsened\n");
        return EXIT_FAILURE;
    }

    const char* matrix_file = argv[1];        ///< Path to Matrix Market file containing sparse matrix
    const char* modes_string = NULL;          ///< SpMV implementation modes (comma-separated)
    const char* output_format = "human";      ///< Output format for metrics (default: human-readable)
    const char* output_file = NULL;           ///< Output file for metrics (default: stdout)

    // Parse command-line arguments to find mode, output format, and output file
    for (int i = 2; i < argc; ++i) {
        if (strncmp(argv[i], "--mode=", 7) == 0) {
            modes_string = argv[i] + 7;  // Get modes value after "--mode="
        } else if (strncmp(argv[i], "--output-format=", 16) == 0) {
            output_format = argv[i] + 16;  // Get format value after "--output-format="
        } else if (strncmp(argv[i], "--output-file=", 14) == 0) {
            output_file = argv[i] + 14;  // Get filename after "--output-file="
        }
    }

    // Validate that modes were specified
    if (modes_string == NULL) {
        fprintf(stderr, "Error: mode not specified. Use --mode=<mode1[,mode2,...]>\n");
        return EXIT_FAILURE;
    }

    // Validate output format
    if (strcmp(output_format, "human") != 0 && strcmp(output_format, "json") != 0 && strcmp(output_format, "csv") != 0) {
        fprintf(stderr, "Error: Invalid output format '%s'. Use --output-format=<human|json|csv>\n", output_format);
        return EXIT_FAILURE;
    }

    // Parse modes (split by comma) BEFORE loading matrix
    char modes_buffer[256];
    strncpy(modes_buffer, modes_string, sizeof(modes_buffer) - 1);
    modes_buffer[sizeof(modes_buffer) - 1] = '\0';
    
    const char* mode_tokens[10];  // Support up to 10 modes
    int num_modes = 0;
    
    char* token = strtok(modes_buffer, ",");
    while (token != NULL && num_modes < 10) {
        mode_tokens[num_modes++] = token;
        token = strtok(NULL, ",");
    }
    
    // Validate all modes BEFORE loading matrix (saves time on invalid modes)
    printf("Validating %d mode(s): ", num_modes);
    for (int i = 0; i < num_modes; i++) {
        printf("%s%s", mode_tokens[i], (i < num_modes - 1) ? ", " : "\n");
        
        SpmvOperator* op = get_operator(mode_tokens[i]);
        if (op == NULL) {
            fprintf(stderr, "Error: Unknown mode '%s'\n", mode_tokens[i]);
            fprintf(stderr, "Available modes: csr, ellpack-naive, ellpack, stencil5, stencil5-opt, stencil5-shared, stencil5-coarsened, stencil5-mgpu, stencil5-no-colindices, stencil5-no-colindices-opt, amgx-stencil\n");
            return EXIT_FAILURE;
        }
    }

    // Load the matrix from Matrix Market file into a generic structure (AFTER mode validation)
    printf("\nLoading matrix: %s\n", matrix_file);
    MatrixData mat; ///< Container for matrix data loaded from file
    if (load_matrix_market(matrix_file, &mat) != 0) {
        fprintf(stderr, "Failed to load matrix %s\n", matrix_file);
        return EXIT_FAILURE;
    }

    printf("Matrix loaded: %d rows, %d cols, %d nonzeros\n", mat.rows, mat.cols, mat.nnz);
    printf("Testing %d mode(s): ", num_modes);
    for (int i = 0; i < num_modes; i++) {
        printf("%s%s", mode_tokens[i], (i < num_modes - 1) ? ", " : "\n");
    }

    // Allocate and initialize input/output vectors on the host (shared across modes)
    double* x = (double*)malloc(mat.cols * sizeof(double)); ///< Input vector for SpMV operation (x in y = A*x)
    double* y = (double*)malloc(mat.rows * sizeof(double)); ///< Output vector for SpMV operation (y in y = A*x)
    if (!x || !y) {
        fprintf(stderr, "Error allocating vectors\n");
        return EXIT_FAILURE;
    }

    // Initialize vectors with appropriate values
    for (int i = 0; i < mat.cols; i++) x[i] = 1.0; // Fill input vector with 1.0
    
    // Loop through all requested modes
    for (int mode_idx = 0; mode_idx < num_modes; mode_idx++) {
        const char* current_mode = mode_tokens[mode_idx];
        
        printf("\n=== Testing mode: %s ===\n", current_mode);
        
        // Select the corresponding SpMV operator (already validated)
        SpmvOperator* op = get_operator(current_mode);
        
        // Initialize the SpMV operator (ELLPACK reused if already built)
        if (op->init(&mat) != 0) {
            fprintf(stderr, "Failed to initialize operator '%s'\n", op->name);
            continue;
        }
        
        // Reset output vector for this mode
        memset(y, 0, mat.rows * sizeof(double));
        
        // Statistical benchmark with outlier detection
        printf("Running statistical benchmark (10 iterations)...\n");
        BenchmarkStats bench_stats;
        if (benchmark_with_stats(op->run_timed, x, y, 10, &bench_stats) != 0) {
            fprintf(stderr, "Statistical benchmark failed for mode '%s'\n", op->name);
            op->free();
            continue;
        }
        
        printf("Completed: %d valid runs, %d outliers removed\n", 
               bench_stats.valid_runs, bench_stats.outliers_removed);
        
        // Calculate performance metrics using median time
        BenchmarkMetrics metrics;
        calculate_spmv_metrics(bench_stats.median_ms, &mat, op->name, &metrics);
        
        // Add GPU specifications to metrics
        if (get_gpu_properties(&metrics) != 0) {
            fprintf(stderr, "Warning: Could not retrieve GPU properties\n");
        }
        
        // Output metrics (for multi-mode: human to stdout, for single-mode: support file output)
        if (num_modes == 1) {
            // Single mode: support file output as before
            FILE* output_fp = stdout;
            if (output_file != NULL) {
                output_fp = fopen(output_file, "w");
                if (output_fp == NULL) {
                    fprintf(stderr, "Error: Could not open output file '%s' for writing\n", output_file);
                    op->free();
                    continue;
                }
                printf("Writing metrics to file: %s\n", output_file);
            }
            
            if (strcmp(output_format, "json") == 0) {
                print_metrics_json(&metrics, output_fp);
            } else if (strcmp(output_format, "csv") == 0) {
                print_metrics_csv(&metrics, output_fp);
            } else {
                print_benchmark_metrics(&metrics, output_fp);
            }
            
            if (output_fp != stdout) {
                fclose(output_fp);
                printf("Metrics successfully written to: %s\n", output_file);
            }
        } else {
            // Multi-mode: human output to stdout for comparison
            print_benchmark_metrics(&metrics, stdout);
        }
        
        printf("SpMV completed successfully using mode: %s\n", op->name);
        
        // Free GPU memory after each mode to prevent accumulation on large matrices
        op->free();
    }
    
    printf("\n=== Multi-mode benchmark completed ===\n");

    // Free host memory for vectors and matrix data
    free(x);
    free(y);
    
    // Free matrix entries allocated in load_matrix_market
    if (mat.entries) {
        free(mat.entries);
    }

    return EXIT_SUCCESS;
}
