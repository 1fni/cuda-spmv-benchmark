#include "hip/hip_runtime.h"
/**
 * @file spmv_stencil.cu
 * @brief Implements SpMV using custom CUDA kernels optimized for 5-point stencil matrices.
 *
 * @details
 * This file provides specialized SpMV implementation for 5-point stencil patterns commonly
 * found in finite difference computations. The implementation uses ELLPACK format converted
 * from CSR to optimize memory access patterns on GPU architectures:
 * - Converts CSR matrix to ELLPACK format for better memory coalescing
 * - Provides two kernel variants: generic and pattern-optimized for diffusion problems
 * - Implements timing measurements using CUDA events
 * - Handles both interior points (regular 5-point pattern) and boundary conditions
 *
 * Author: Bouhrour Stephane
 * Date: 2025-07-15
 */

#include <stdio.h>
#include "spmv.h"
#include "io.h"

ELLPACKMatrix ellpack_matrix; ///< Global ELLPACK matrix structure used by stencil operator

// GPU device memory pointers
static double *d_values = nullptr;  ///< Device memory for ELLPACK matrix values
static int *d_indices = nullptr;    ///< Device memory for ELLPACK column indices  
static double *dX = nullptr;        ///< Device memory for input vector x
static double *dY = nullptr;        ///< Device memory for output vector y

// SpMV computation parameters
static const double alpha = 1.0;    ///< Alpha coefficient for SpMV operation (y = alpha*A*x + beta*y)
static const double beta = 0.0;     ///< Beta coefficient for SpMV operation (y = alpha*A*x + beta*y)

/**
 * @brief CUDA kernel for SpMV optimized for 5-point stencil patterns with separate handling for interior and boundary points.
 * @details This kernel distinguishes between interior grid points (which follow regular 5-point stencil pattern)
 * and boundary/corner points (which require general ELLPACK processing). Interior points use direct indexing
 * for optimal performance, while boundary points use loop-based processing.
 * @param data ELLPACK matrix values array
 * @param col_indices ELLPACK column indices array  
 * @param vec Input vector x
 * @param result Output vector y (result of A*x)
 * @param num_rows Number of matrix rows
 * @param max_nonzero_per_row Maximum non-zeros per row (ELLPACK width)
 * @param alpha Scalar multiplier for matrix-vector product
 * @param beta Scalar multiplier for existing result vector (not used in current implementation)
 */
__global__ void ellpack_matvec_optimized_diffusion_pattern_middle_and_else(const double * data, const int* col_indices, const double * vec, double * result, int num_rows, int max_nonzero_per_row, const double alpha,  const double beta, int grid_size) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < num_rows){

		// Separate the interior (middle) stencil convolution from borders/corners:
		// on sufficiently large grids the interior pattern represents by far the heaviest workload,
		// and handling it separately minimizes warp divergence.
		// Further splitting the minor patterns (corners/boundaries) yields negligible benefit.
		// Follow logic with grid flatten and represent by x vector
		// Where the sparse matrice stores the convolution pattern value 

		// Applies the 5-point stencil to interior rows of a structured grid, performing coalesced memory accesses.
		// The interior (middle) stencil convolution is isolated from borders and corners because on sufficiently
		// large grids it represents by far the heaviest workload; handling it separately minimizes warp divergence
		// on the critical path. Further subdividing the minor border/corner patterns yields negligible performance benefit.
		
		// Convert 1D row index to 2D grid coordinates
		int i = row / grid_size;  // row in 2D grid
		int j = row % grid_size;  // column in 2D grid
		
		// Check if interior point (not on boundaries)
		if (i > 0 && i < grid_size-1 && j > 0 && j < grid_size-1) {
																//if ((row < num_rows - num_rows) && (row > num_rows- 1) && (row % num_rows!= 0) && (row % (num_rows- 1) != 0)) {//milieu
																//if ((row < num_rows - size_arcfile) && (row > size_arcfile - 1) && (row % size_arcfile != 0) && (row % (size_arcfile - 1) != 0)) {//milieu
			double sum = 0.0f;
			int offset = row * max_nonzero_per_row;
			sum += data[offset] * vec[row - grid_size];  // North neighbor
			sum += data[offset + 1] * vec[row - 1];      // West neighbor
			sum += data[offset + 2] * vec[row];          // Center point
			sum += data[offset + 3] * vec[row + 1];      // East neighbor
			sum += data[offset + 4] * vec[row + grid_size]; // South neighbor
			result[row] = alpha*sum;
			//printf("row %d %lf %lf, %lf %lf, %lf %lf, %lf %lf, %lf %lf\n", row, data[offset] , vec[row - size_grid],data[offset+1] , vec[row - 1],data[offset+2] , vec[row],data[offset+3] , vec[row + 1],data[offset+4] , vec[row + size_grid]);
		}
		else{
			//else if((row == 0) || (row == num_rows- 1) || (row == num_rows - num_rows) || ((row == num_rows - 1)) || ((row < num_rows- 1) && (row > 0)) || ((row > (num_rows - num_rows)) && (row < (num_rows - 1))) || ((row != 0) && (row != (num_rows - num_rows)) && ((row % num_rows) == 0)) || ((row != (num_rows- 1)) && (row != (num_rows - 1)) && ((row % (num_rows- 1)) == 0)) ) {//others, edge and corners
			//else if((row == 0) || (row == size_arcfile - 1) || (row == num_rows - size_arcfile) || ((row == num_rows - 1)) || ((row < size_arcfile - 1) && (row > 0)) || ((row > (num_rows - size_arcfile)) && (row < (num_rows - 1))) || ((row != 0) && (row != (num_rows - size_arcfile)) && ((row % size_arcfile) == 0)) || ((row != (size_arcfile - 1)) && (row != (num_rows - 1)) && ((row % (size_arcfile - 1)) == 0)) ) {//others, edge and corners
			//printf("elserow %d\n", row);
			double sum = 0.0f;
			int offset = row * max_nonzero_per_row;

#pragma unroll
			for (int i = 0; i < max_nonzero_per_row; ++i) {
				int col = col_indices[row * max_nonzero_per_row + i];
				if (col >= 0) {  // Vérifie si l'indice de colonne est valide
					sum += data[offset + i] * vec[col];
				}
			}

			// Stocke le résultat final
			result[row] = alpha*sum;
		}
		}
		}

/**
 * @brief Generic CUDA kernel for ELLPACK SpMV computation.
 * @details This kernel performs standard ELLPACK format SpMV using loop-based processing
 * for all matrix rows. Each thread processes one matrix row by iterating through its
 * non-zero elements stored in ELLPACK format.
 * @param data ELLPACK matrix values array
 * @param col_indices ELLPACK column indices array
 * @param vec Input vector x  
 * @param result Output vector y (result of A*x)
 * @param num_rows Number of matrix rows
 * @param max_nonzero_per_row Maximum non-zeros per row (ELLPACK width)
 * @param alpha Scalar multiplier for matrix-vector product
 * @param beta Scalar multiplier for existing result vector
 */
__global__ void ellpack_matvec_optimized_diffusion(const double * data, const int* col_indices, const double * vec, double * result, int num_rows, int max_nonzero_per_row, const double alpha,  const double beta) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	// Vérifie si le thread traite une ligne valide
	if (row < num_rows) {
		double sum = 0.0f;

		// Utilise des accès directs au lieu d'une boucle pour les colonnes non nulles
#pragma unroll
		for (int i = 0; i < max_nonzero_per_row; ++i) {
			int col = col_indices[row * max_nonzero_per_row + i];
			if (col >= 0) {  // Vérifie si l'indice de colonne est valide
				sum += alpha * data[row * max_nonzero_per_row + i] + beta * vec[col];
			}
		}

		// Stocke le résultat final
		result[row] = sum;
	}
}


/**
 * @brief Converts CSR matrix format to ELLPACK format for optimized GPU processing.
 * @details This function transforms a CSR (Compressed Sparse Row) matrix into ELLPACK format
 * by determining the maximum row width and creating padded arrays for values and indices.
 * ELLPACK format enables better memory coalescing on GPU by storing matrix data in
 * row-major order with uniform row lengths.
 * @param csr_matrix Pointer to the source CSR matrix structure
 * @return int 0 on success, non-zero on failure
 */
int build_ellpack_from_csr_struct(CSRMatrix *csr_matrix){
	int max_nonzeros = 0;
	for (int i = 0; i < csr_matrix->nb_rows; ++i) {
		int row_nonzeros = csr_matrix->row_ptr[i + 1] - csr_matrix->row_ptr[i];
		if (row_nonzeros > max_nonzeros) {
			max_nonzeros = row_nonzeros;
		}
	}
	ellpack_matrix.ell_width = (max_nonzeros > MAX_WIDTH) ? MAX_WIDTH : max_nonzeros;
	printf("ELL WIDTH %d\n", ellpack_matrix.ell_width);

	ellpack_matrix.nb_rows = csr_matrix->nb_rows;
	ellpack_matrix.nb_cols = csr_matrix->nb_cols;
	ellpack_matrix.nb_nonzeros = csr_matrix->nb_nonzeros;
	// grid_size will be assigned in stencil5_init

	int total_ell_elements = csr_matrix->nb_rows * ellpack_matrix.ell_width;
	ellpack_matrix.indices = (int *)calloc(total_ell_elements, sizeof(int));
	ellpack_matrix.values = (double *)calloc(total_ell_elements, sizeof(double));

	// Debug: afficher la structure CSR avant conversion
	//printf("CSR Matrix debug:\n");
	//for (int i = 0; i < csr_matrix->nb_rows; ++i) {
	//	printf("Row %d CSR: ", i);
	//	for (int j = csr_matrix->row_ptr[i]; j < csr_matrix->row_ptr[i + 1]; ++j) {
	//		printf("(col:%d val:%lf) ", csr_matrix->col_indices[j], csr_matrix->values[j]);
	//	}
	//	printf("\n");
	//}
	
	//populate ELLPACK format
	for (int i = 0; i < csr_matrix->nb_rows; ++i) {
		int ell_index = 0;
		for (int j = csr_matrix->row_ptr[i]; j < csr_matrix->row_ptr[i + 1]; ++j) {
			if (ell_index < ellpack_matrix.ell_width) {
				ellpack_matrix.indices[i * ellpack_matrix.ell_width + ell_index] = csr_matrix->col_indices[j];
				ellpack_matrix.values[i * ellpack_matrix.ell_width + ell_index] = csr_matrix->values[j];
				ell_index++;
			} else {
				break;
			}
		}
	}
	// Debug: afficher la conversion pour toutes les lignes
	//for (int i = 0; i < ellpack_matrix.nb_rows; i++) {
	//	printf("Row %d ELLPACK: ", i);
	//	for (int j = 0; j < ellpack_matrix.ell_width; j++) {
	//		printf("(col:%d val:%lf) ", ellpack_matrix.indices[ellpack_matrix.ell_width*i + j], ellpack_matrix.values[ellpack_matrix.ell_width*i + j]);
	//	}
	//	printf("\n");
	//}
	return 0;
}

/**
 * @brief Initializes the stencil5 SpMV operator with matrix data and GPU memory allocation.
 * @details This function performs the complete initialization sequence for stencil-based SpMV:
 * 1. Converts input MatrixData to CSR format using global csr_mat structure
 * 2. Converts CSR to ELLPACK format for optimized GPU access patterns
 * 3. Allocates GPU device memory for matrix values, indices, and input/output vectors
 * 4. Transfers matrix data from host to device memory
 * @param mat Pointer to MatrixData structure containing the sparse matrix
 * @return int 0 on successful initialization, non-zero on failure
 */
int stencil5_init(MatrixData* mat) {

	//build CSR from MatrixData* mat then convert in ELLPACK
	build_csr_struct(mat);
	build_ellpack_from_csr_struct(&csr_mat);
	
	// Store grid_size from MatrixData for stencil operations
	ellpack_matrix.grid_size = mat->grid_size;

	size_t size_values = ellpack_matrix.nb_rows * ellpack_matrix.ell_width * sizeof(double);
	size_t size_indices = ellpack_matrix.nb_rows * ellpack_matrix.ell_width * sizeof(int);
	size_t size_vec = ellpack_matrix.nb_rows * sizeof(double);

	// Allocation GPU
	CUDA_CHECK(hipMalloc((void**)&d_values, size_values));
	CUDA_CHECK(hipMalloc((void**)&d_indices, size_indices));
	CUDA_CHECK(hipMalloc((void**)&dX, size_vec));
	CUDA_CHECK(hipMalloc((void**)&dY, size_vec));

	// Transfert H2D
	CUDA_CHECK(hipMemcpy(d_values, ellpack_matrix.values, size_values, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_indices, ellpack_matrix.indices, size_indices, hipMemcpyHostToDevice));
	
	// Synchronisation pour s'assurer que le transfert est terminé
	CUDA_CHECK(hipDeviceSynchronize());
	
	// Debug: vérifier les données sur GPU après transfert
	//printf("Verification GPU data after H2D transfer:\n");
	//double *h_verify_values = (double*)malloc(size_values);
	//int *h_verify_indices = (int*)malloc(size_indices);
	//
	//CUDA_CHECK(hipMemcpy(h_verify_values, d_values, size_values, hipMemcpyDeviceToHost));
	//CUDA_CHECK(hipMemcpy(h_verify_indices, d_indices, size_indices, hipMemcpyDeviceToHost));
	//
	//for (int i = 0; i < ellpack_matrix.nb_rows; i++) {
	//	printf("GPU Row %d: ", i);
	//	for (int j = 0; j < ellpack_matrix.ell_width; j++) {
	//		int idx = i * ellpack_matrix.ell_width + j;
	//		printf("(col:%d val:%lf) ", h_verify_indices[idx], h_verify_values[idx]);
	//	}
	//	printf("\n");
	//}
	//
	//free(h_verify_values);
	//free(h_verify_indices);
	return 0;
}

/**
 * @brief Executes the stencil5 SpMV computation with precise kernel timing measurement.
 * @details This function performs the complete SpMV execution workflow:
 * 1. Transfers input vector from host to device memory
 * 2. Launches optimized CUDA kernel for 5-point stencil pattern
 * 3. Measures precise kernel execution time using CUDA events
 * 4. Transfers result vector back to host memory
 * 5. Computes and displays checksum for verification
 * @param x Input vector (host memory)
 * @param y Output vector (host memory) - will contain result of A*x
 * @param kernel_time_ms Output parameter for kernel execution time in milliseconds
 * @return int 0 on successful execution, non-zero on failure
 */
int stencil5_run_timed(const double* x, double* y, double* kernel_time_ms) {
	size_t size_vec = ellpack_matrix.nb_rows * sizeof(double);
	
	CUDA_CHECK(hipMemcpy(dX, x, size_vec, hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Kernel SpMV
	hipEventRecord(start);
	int threads = 32;
	int blocks = (ellpack_matrix.nb_rows + threads - 1) / threads;
	//ellpack_matvec_optimized_diffusion<<<blocks, threads>>>(d_values, d_indices, dX, dY,ellpack_matrix.nb_rows, ellpack_matrix.ell_width, alpha, beta);
	printf("Matrix rows: %d, Grid size: %d\n", ellpack_matrix.nb_rows, ellpack_matrix.grid_size);
	ellpack_matvec_optimized_diffusion_pattern_middle_and_else<<<blocks, threads>>>(d_values, d_indices, dX, dY,ellpack_matrix.nb_rows, ellpack_matrix.ell_width, alpha, beta, ellpack_matrix.grid_size);

	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float computeTime;
	hipEventElapsedTime(&computeTime, start, stop);
	
	// Return precise kernel timing for metrics calculation
	*kernel_time_ms = (double)computeTime;
	printf("[Stencil5] Kernel time: %.3f ms\n", computeTime);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy result
	CUDA_CHECK(hipMemcpy(y, dY, size_vec, hipMemcpyDeviceToHost));

	// Print result
	printf("Result:\n");
	double check_sum = 0.0;
	for (int i = 0; i < ellpack_matrix.nb_rows; i++) {
		check_sum += y[i];
	}
	printf("check_sum %le\n", check_sum);

	return 0;
}

/**
 * @brief Frees all GPU device memory allocated by the stencil5 operator.
 * @details This cleanup function releases all CUDA device memory allocations:
 * - Matrix values and indices arrays
 * - Input and output vector device memory
 * Should be called when the stencil5 operator is no longer needed.
 */
void stencil5_free() {
	printf("[STENCIL5] Cleaning up\n");
	
	// Free GPU memory  
	CUDA_CHECK(hipFree(d_values));
	CUDA_CHECK(hipFree(d_indices));
	CUDA_CHECK(hipFree(dX));
	CUDA_CHECK(hipFree(dY));
	
	// Free host ELLPACK arrays
	if (ellpack_matrix.indices) {
		free(ellpack_matrix.indices);
		ellpack_matrix.indices = NULL;
	}
	if (ellpack_matrix.values) {
		free(ellpack_matrix.values);
		ellpack_matrix.values = NULL;
	}
	
	// Free host CSR arrays used to build ELLPACK
	if (csr_mat.row_ptr) {
		free(csr_mat.row_ptr);
		csr_mat.row_ptr = NULL;
	}
	if (csr_mat.col_indices) {
		free(csr_mat.col_indices);
		csr_mat.col_indices = NULL;
	}
	if (csr_mat.values) {
		free(csr_mat.values);
		csr_mat.values = NULL;
	}
}

/**
 * @brief Global SpmvOperator structure for stencil5 implementation.
 * @details This operator provides the interface for 5-point stencil SpMV computations:
 * - name: "stencil5" identifier for operator selection
 * - init: stencil5_init function for GPU memory setup and matrix conversion
 * - run: stencil5_run function for kernel execution and timing
 * - free: stencil5_free function for resource cleanup
 */
SpmvOperator SPMV_STENCIL5 = {
	.name = "stencil5",
	.init = stencil5_init,
	.run_timed = stencil5_run_timed,
	.free = stencil5_free
};

