/**
 * @file spmv_cusparse_csr.cu
 * @brief Implements SpMV (Sparse Matrix-Vector multiplication) using cuSPARSE CSR format.
 *
 * @details
 * Responsibilities:
 *  - Allocate device memory for CSR components (row_ptr, col_indices, values)
 *  - Create cuSPARSE descriptors for CSR matrix and dense vectors
 *  - Perform SpMV using hipsparseSpMV()
 *  - Measure kernel execution time
 *  - Free GPU and cuSPARSE resources
 *
 * Author: Bouhrour Stephane
 * Date: 2025-07-15
 */

#include <stdio.h>
#include <hipsparse.h>
#include "spmv.h"
#include "io.h"

/// Host-side CSR matrix structure holding row_ptr, col_indices, values, and dimensions
CSRMatrix csr_mat; ///< CSR matrix used by GPU routines

/// cuSPARSE handle for library context
static hipsparseHandle_t handle = nullptr; ///< cuSPARSE context handle

/// cuSPARSE descriptors for sparse matrix A and dense vectors X, Y
static hipsparseSpMatDescr_t matA; ///< Descriptor for CSR sparse matrix
static hipsparseDnVecDescr_t vecX, vecY; ///< Descriptors for dense vectors

/// Device workspace buffer for cuSPARSE operations
static void* dBuffer = nullptr;       ///< Pointer to temporary workspace buffer
static size_t bufferSize = 0;         ///< Size of workspace buffer in bytes

/// Device pointers for CSR data and input/output vectors
static int    *dA_csrOffsets = nullptr; ///< Device array of row pointers (CSR)
static int    *dA_columns    = nullptr; ///< Device array of column indices
static double *dA_values     = nullptr; ///< Device array of non-zero values
static double *dX            = nullptr; ///< Device input vector X
static double *dY            = nullptr; ///< Device output vector Y

/// Scalars for SpMV: Y = alpha * A * X + beta * Y
static double alpha = 1.0;           ///< Weight for A*X
static double beta  = 0.0;           ///< Weight for existing Y

/**
 * @brief Builds CSR arrays from host-side MatrixData entries.
 * @details
 * Converts COO-like format (mat->entries) into CSR storage:
 *   - row_ptr: (rows + 1) length, prefix-sums of non-zero counts per row
 *   - col_indices: flattened column indices
 *   - values: flattened non-zero values
 * Allocates and populates host memory, then stores into global csr_mat.
 * @param mat Pointer to MatrixData with fields:
 *            - mat->rows    : number of rows
 *            - mat->cols    : number of columns
 *            - mat->nnz     : number of non-zero entries
 *            - mat->entries : array of {row, col, value} entries (length nnz)
 * @return EXIT_SUCCESS (0) on success, EXIT_FAILURE (non-zero) on allocation error.
 */
int build_csr_struct(MatrixData* mat)
{
    // Allocate row pointer array
    int *row_ptr = (int*)calloc(mat->rows + 1, sizeof(int));
    if (!row_ptr) {
        fprintf(stderr, "[ERROR] calloc failed for row_ptr\n");
        return EXIT_FAILURE;
    }

    // Count non-zeros per row
    for (int i = 0; i < mat->nnz; ++i) {
        int r = mat->entries[i].row;
        row_ptr[r + 1]++;
    }

    // Build prefix sum for row offsets
    for (int i = 1; i <= mat->rows; ++i) {
        row_ptr[i] += row_ptr[i - 1];
    }

    // Allocate column indices and values arrays
    int *col_indices = (int*)malloc(mat->nnz * sizeof(int));
    if (!col_indices) { free(row_ptr); return EXIT_FAILURE; }
    double *values   = (double*)malloc(mat->nnz * sizeof(double));
    if (!values) { free(row_ptr); free(col_indices); return EXIT_FAILURE; }

    // Temporary counters per row
    int *local_count = (int*)calloc(mat->rows, sizeof(int));
    if (!local_count) { free(row_ptr); free(col_indices); free(values); return EXIT_FAILURE; }

    // Populate CSR arrays
    for (int i = 0; i < mat->nnz; ++i) {
        int r = mat->entries[i].row;
        int dst = row_ptr[r] + local_count[r]++;
        col_indices[dst] = mat->entries[i].col;
        values[dst]      = mat->entries[i].value;
    }

    // Store into global CSRMatrix
    csr_mat.row_ptr     = row_ptr;
    csr_mat.col_indices = col_indices;
    csr_mat.values      = values;
    csr_mat.nb_rows     = mat->rows;
    csr_mat.nb_cols     = mat->cols;
    csr_mat.nb_nonzeros = mat->nnz;

    free(local_count);
    return EXIT_SUCCESS;
}

/**
 * @brief Initializes cuSPARSE and allocates GPU memory for CSR SpMV.
 * @details
 *   - Builds host CSR arrays via build_csr_struct()
 *   - Creates cuSPARSE handle and descriptors
 *   - Allocates and copies CSR arrays and vectors to GPU
 *   - Queries and allocates workspace buffer
 * @param mat Pointer to MatrixData for initialization
 * @return EXIT_SUCCESS on success, EXIT_FAILURE on error
 */
int csr_init(MatrixData* mat)
{
    if (build_csr_struct(mat) != EXIT_SUCCESS) return EXIT_FAILURE;

    // Create cuSPARSE context
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate GPU memory for CSR arrays and vectors
    CUDA_CHECK(hipMalloc((void**)&dA_csrOffsets, (csr_mat.nb_rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dA_columns,    csr_mat.nb_nonzeros * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dA_values,     csr_mat.nb_nonzeros * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&dX,            csr_mat.nb_cols     * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&dY,            csr_mat.nb_rows     * sizeof(double)));

    // Copy CSR data to GPU
    CUDA_CHECK(hipMemcpy(dA_csrOffsets, csr_mat.row_ptr,
                          (csr_mat.nb_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dA_columns,    csr_mat.col_indices,
                          csr_mat.nb_nonzeros  * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dA_values,     csr_mat.values,
                          csr_mat.nb_nonzeros  * sizeof(double), hipMemcpyHostToDevice));

    // Create sparse matrix descriptor
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
        csr_mat.nb_rows, csr_mat.nb_cols, csr_mat.nb_nonzeros,
        dA_csrOffsets, dA_columns, dA_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // Create dense vector descriptors
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, csr_mat.nb_cols, dX, HIP_R_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, csr_mat.nb_rows, dY, HIP_R_64F));

    // Query workspace size and allocate
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY,
        HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSize));
    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    return EXIT_SUCCESS;
}

/**
 * @brief Executes CSR SpMV kernel and reports result.
 * @details
 *   - Copies input vector to GPU
 *   - Launches hipsparseSpMV and measures execution time
 *   - Copies output vector back to host and computes checksum
 * @param x Host input vector (length nb_cols)
 * @param y Host output vector (length nb_rows)
 * @return EXIT_SUCCESS on success
 */
int csr_run(const double* x, double* y)
{
    // Copy input vector
    CUDA_CHECK(hipMemcpy(dX, x, csr_mat.nb_cols * sizeof(double), hipMemcpyHostToDevice));

    // Record and perform SpMV
    hipEvent_t start, stop;
    float time_ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY,
        HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);
    printf("[cuSPARSE CSR] Compute SpMV: %.3f ms\n", time_ms);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy and verify result
    CUDA_CHECK(hipMemcpy(y, dY, csr_mat.nb_rows * sizeof(double), hipMemcpyDeviceToHost));
    double checksum = 0.0;
    for (int i = 0; i < csr_mat.nb_rows; ++i) checksum += y[i];
    printf("[CSR] checksum: %le\n", checksum);

    return EXIT_SUCCESS;
}

/**
 * @brief Frees GPU memory and destroys cuSPARSE resources.
 * @details
 *   - Releases device buffers and workspace
 *   - Destroys cuSPARSE descriptors and handle
 */
void csr_free()
{
    printf("[CSR] Cleaning up\n");
    
    // Free GPU memory
    CUDA_CHECK(hipFree(dA_values));
    CUDA_CHECK(hipFree(dA_columns));
    CUDA_CHECK(hipFree(dA_csrOffsets));
    CUDA_CHECK(hipFree(dX));
    CUDA_CHECK(hipFree(dY));
    CUDA_CHECK(hipFree(dBuffer));

    // Free cuSPARSE objects
    if (vecX) hipsparseDestroyDnVec(vecX);
    if (vecY) hipsparseDestroyDnVec(vecY);
    if (matA) hipsparseDestroySpMat(matA);
    if (handle) hipsparseDestroy(handle);
    
    // Free host CSR arrays
    if (csr_mat.row_ptr) {
        free(csr_mat.row_ptr);
        csr_mat.row_ptr = NULL;
    }
    if (csr_mat.col_indices) {
        free(csr_mat.col_indices);
        csr_mat.col_indices = NULL;
    }
    if (csr_mat.values) {
        free(csr_mat.values);
        csr_mat.values = NULL;
    }
}

/**
 * @brief Registers the CSR SpMV operator in the benchmark suite.
 * @details
 *   Provides function pointers for init, run, and free routines.
 */
SpmvOperator SPMV_CSR = {
    .name = "csr",
    .init = csr_init,
    .run  = csr_run,
    .free = csr_free
};
